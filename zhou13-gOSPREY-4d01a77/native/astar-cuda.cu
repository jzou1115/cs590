#include "hip/hip_runtime.h"
/* 
 * If you think this code is a mess... Well, this is originally written in C
 * but then I found that nvcc does not support C99.  So now it is a frank which
 * combines custom either c and cpp.
 *
 * To successor:  Please write detailed comment on function header and global
 * variable!
 */
#define __STDC_LIMIT_MACROS
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include <algorithm>
#include <stdint.h>

#include "pq.h"

#include "astar-cuda.h"

#define nVERBOSE_DEBUG

typedef struct data_t {
    float f_score;
    float g_score;
    int level;
    uchar conf[MAX_LEVEL];
} data_t;

typedef struct heap_t {
    float v;
    int idx;
} heap_t;

bool operator< (const data_t &a, const data_t &b)
{
    return a.f_score < b.f_score;
}

// number of data_t s on GPU
static int data_capacity;
// number of data_t s reserved on GPU before for ``shrink''
static int data_reserved;

// GPU parameter
static int num_block;
static int num_local;
static int num_local2;
static int num_global; // =num_block*num_local
static int heap_capacity;  // =data_capacity/num_global
static __device__ __constant__ int d_heap_capacity;

// priority_queue on CPU to store the result return by GPU
#define heap_cmp(a, b) ((a)->f_score < (b)->f_score)
static priority_queue(data_t *) heap;

// GPU heap, contain point to d_data
// Heap of heap: d_heap[heap_capacity], d_heap[2*heap_capacity], ...
static heap_t *d_heap;
// Memory pool
static data_t *d_data;
// Used by shrink function
static data_t *d_data2;
// Answer is put in this array so that CPU can copy.
static data_t *d_output;

// Used by shrink function for sorting
static float *d_data_val;
// Used by shrink function
static int   *d_data_used;

// d_heap_size[i]: heap size of ith heap
static int *d_heap_size;
// d_parent[i]: the node preparing to extend
static int *d_parent;
// radix sort by level of node to decrease branch divergence
static int *d_radix;
// node we need to calculate energy
static int2 *d_input;
// number of child for d_parent[i].
static int *d_node_cnt;

// these point to a number, not a array
static int *d_data_size;
static int *d_output_size;
static int *d_begin_index;
static int *d_begin_index2;
static uint *d_optimal;

// necessary to compute energy
static int *d_node_offset;
static int *d_rot_per_level;
static float *d_self_energy;
static float *d_reduce_energy;
static float *d_pm_energy;

static int rounds = 0;
static uint optimal = 0;
static int curr_conf[MAX_LEVEL];
static int h_num_child = 0;
static int h_output_size = 0;
static int h_data_size = 0;

static int max_data;
static float throw_min;

// map a float number to a unsigned such that:
//     if a < b, flip_float(a) < flip_float(b)
    __host__ __device__
uint flip_float(float fl)
{
    union {
        float fl;
        int  u;
    } un;
    un.fl = fl;
    return un.u ^ ((un.u >> 31) | 0x80000000);
}

    __attribute__((unused))
float reverse_flip_float(int u)
{
    union {
        float f;
        int u;
    } un;
    un.u = u ^ ((~u >> 31) | 0x80000000);
    return un.f;
}

#define CHK_CUDA(exp) \
    if (1)  { \
        hipError_t v = (exp); \
        if (v != hipSuccess) { \
            fprintf(stderr, "CUDA ERR [File %s, Line: %d]: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(v)); \
            exit(EXIT_FAILURE); \
        } \
    } else


#define copy_from_device(d_ptr0) ({ \
                                  typeof(d_ptr0) d_ptr = d_ptr0; \
                                  typeof(*d_ptr) h; \
                                  CHK_CUDA(hipMemcpy(&h, d_ptr, sizeof(h), hipMemcpyDeviceToHost)); \
                                  h; });

    template <typename T>
inline void copy_to_device(T *d_ptr, const T &val)
{
    CHK_CUDA(hipMemcpy(d_ptr, &val, sizeof(T), hipMemcpyHostToDevice));
}

    template <typename T>
inline void cuda_free(T *&d_ptr)
{
    if (d_ptr)
        CHK_CUDA(hipFree(d_ptr));
    d_ptr = NULL;
}

// compute minimal energy for ``data_t *data''
    __device__
void __compute_score(
        int tree_level,
        int rot_cnt,
        data_t *data,
        float old_g_score,
        const int node_offset[],
        const int rot_per_level[],
        const float self_energy[],
        const float reduce_energy[],
        const float pm_energy[])
{
    int level = data->level;
    float g_score = old_g_score; /* use previous g score */
    float h_score = 0.f;

    /* compute g delta */
    {
        int idx = node_offset[level] + data->conf[level];
        for (int j = 0; j < level; ++j) {
            int cord = idx*rot_cnt + node_offset[j]+data->conf[j];
            g_score += reduce_energy[cord];
        }
        g_score += self_energy[idx];
    }

    /* compute h */
    {
        for (int i = level+1; i < tree_level; ++i) {
            float min_energy = FLT_MAX;
            for (int j = 0; j < rot_per_level[i]; ++j) {
                int idx = node_offset[i]+j;
                float cur_energy = self_energy[idx];

                for (int k = 0; k <= level; ++k) {
                    int cord = (idx*rot_cnt) + (node_offset[k]+data->conf[k]);
                    cur_energy += reduce_energy[cord];
                }
                /*
                   int k;
                   for (k = 0; k+1 <= level; k += 2) {
                   int cord0 = (idx*rot_cnt) + (node_offset[k]+data->conf[k]);
                   int cord1 = (idx*rot_cnt) + (node_offset[k+1]+data->conf[k+1]);
                   t0 += reduce_energy[cord0];
                   t1 += reduce_energy[cord1];
                   }
                   if (k <= level) {
                   int cord0 = (idx*rot_cnt) + (node_offset[k]+data->conf[k]);
                   t0 += reduce_energy[cord0];
                   }
                 */

                int cord = (idx)*(tree_level+1) + i+1;
                cur_energy += pm_energy[cord];
                min_energy = min(min_energy, cur_energy);
            }
            h_score += min_energy;
        }
    }
    data->f_score = g_score + h_score;
    data->g_score = g_score;
}

// initialize everything
__global__
void d_initialize(
        int tree_level,
        int rot_cnt,

        heap_t *g_heap,
        data_t *g_data,
        int  g_heap_size[],
        int *g_data_size,

        uint *g_optimal,

        int *g_output_size,
        int *g_begin_index,
        int *g_begin_index2,

        int g_radix[],

        const int   g_node_offset[],
        const int   g_rot_per_level[],
        const float g_self_energy[],
        const float g_reduce_energy[],
        const float g_pm_energy[])
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int lid = threadIdx.x;

    __shared__ int   node_offset[MAX_LEVEL];
    __shared__ int   rot_per_level[MAX_LEVEL];
    extern __shared__ float self_energy[];

    if (lid < tree_level) {
        node_offset[lid] = g_node_offset[lid];
        rot_per_level[lid] = g_rot_per_level[lid];
    }
    for (int i = id; i < rot_cnt; i += blockDim.x)
        self_energy[i] = g_self_energy[i];

    __syncthreads();

    g_heap_size[id] = 0;

    if (id < rot_per_level[0]) {
        data_t data;
        data.level = 0;
        data.conf[0] = id;
        __compute_score(tree_level,
                        rot_cnt,
                        &data,
                        0.f,
                        node_offset,
                        rot_per_level,
                        self_energy,
                        g_reduce_energy,
                        g_pm_energy);
        g_data[id] = data;

        int index = id*d_heap_capacity + 1;
        g_heap[index].v = data.f_score;
        g_heap[index].idx = id;
        g_heap_size[id] = 1;
        atomicMin(g_optimal, flip_float(data.f_score));

#ifdef VERBOSE_DEBUG
        printf("[%d]init: fscore: %.3f gscore: %.3f\n",
               id, data.f_score, data.g_score);
#endif
    }

    if (id < tree_level)
        g_radix[id] = 0;

    if (id == 0) {
        *g_data_size = rot_per_level[0];
        *g_output_size = *g_begin_index = *g_begin_index2 = 0;
    }
}

// extract the minimal element from heap
__global__ void d_delete_min(
        int tree_level,
        int num_child,

        heap_t g_heap[],
        data_t g_data[],
        data_t g_output[],

        int  g_heap_size[],
        int *g_data_size,
        int *g_output_size,

        int *g_begin_index,
        int *g_begin_index2,

        uint *g_optimal,

        int  g_parent[],
        int  g_radix[],
        int  g_node_cnt[],

        const int g_rot_per_level[])
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int lid = threadIdx.x;

    int heap_size = g_heap_size[id];
    bool in_work = (heap_size != 0);

    /* =========================== delete_min ============================ */
    heap_t *heap = g_heap + id*d_heap_capacity;

    float node_score;
    int node_index = -1;
    int cur_level;

    if (in_work) {
        node_score = heap[1].v;
        node_index = heap[1].idx;
        cur_level  = g_data[node_index].level + 1;

#ifdef VERBOSE_DEBUG
        printf("[%d]pop node: %d:%.3f\n", id, node_index, heap[1].v);
#endif

        heap_t now_val = heap[heap_size--];
        g_heap_size[id] = heap_size;

        /* pop from heap */
        int now = 1;
        int next;
        while ((next = now*2) <= heap_size) {
            heap_t next_val = heap[next];
            heap_t next_val2 = heap[next+1];
            bool inc = (next+1 <= heap_size) && (next_val2.v < next_val.v);
            if (inc) {
                next += 1;
                next_val = next_val2;
            }

            if (next_val.v < now_val.v) {
                heap[now] = next_val;
                now = next;
            } else
                break;
        }
        heap[now] = now_val;
    }

    __shared__ int  rot_per_level[MAX_LEVEL];
    __shared__ int  l_radix[MAX_LEVEL];
    __shared__ uint l_optimal;
    l_optimal = UINT_MAX;
    if (lid < tree_level) {
        l_radix[lid] = 0;
        rot_per_level[lid] = g_rot_per_level[lid];
    }
    __syncthreads();

    /* =========================== update answer ============================ */
    if (in_work) {
        if (cur_level == tree_level) {
            /* add to answer array if we are leaves */
            int idx = atomicAdd(g_output_size, 1);
#ifdef VERBOSE_DEBUG
            printf("   >>   puts %d(%.8f, %.8f) on %d\n", node_index, node_score, g_data[node_index].f_score, idx);
#endif
            g_output[idx] = g_data[node_index];
            if (heap_size > 0)
                atomicMin(&l_optimal, flip_float(heap[1].v));

            in_work = false;
            node_index = -1;
        } else
            atomicMin(&l_optimal, flip_float(node_score));
    }
    __syncthreads();
    if (lid == 0) {
        atomicMin(g_optimal, l_optimal);
    }
    /* ======================== radix sort first part ======================== */
    g_parent[id] = node_index;

    if (in_work)
        atomicAdd(&l_radix[cur_level], rot_per_level[cur_level]);
    __syncthreads();
    if (lid < tree_level) {
        g_node_cnt[id] = l_radix[lid];
    }

    for (int i = 1; i < tree_level; i *= 2) {
        if (lid - i >= 0 && lid < tree_level)
            l_radix[lid] += l_radix[lid - i];
        __syncthreads();
    }
    if (lid < tree_level) {
        atomicAdd(&g_radix[lid], l_radix[lid]);
    }

    if (id == 0) {
        *g_data_size += num_child;
        *g_begin_index = *g_begin_index2;
    }
}

// sort extracted node by their level to decreasing branch divergance
__global__ void d_radix_sort(
        int tree_level,
        data_t g_data[],

        int *g_data_size,

        int g_radix[],
        int g_parent[],
        int g_node_cnt[],

        int2 g_input[],

        const int g_rot_per_level[])
{
    __shared__ int rot_per_level[MAX_LEVEL];
    __shared__ int l_radix[MAX_LEVEL];

    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int lid = threadIdx.x;

    if (lid < tree_level) {
        rot_per_level[lid] = g_rot_per_level[lid];

        int t = g_node_cnt[id];
        l_radix[lid] = atomicSub(&g_radix[lid], t) - t;
    }
    __syncthreads();

    int node_index = g_parent[id];
#ifdef VERBOSE_DEBUG
    printf("[%d]radix sort: node_index: %d\n", id, node_index);
#endif

    if (node_index >= 0) {
        int cur_level = g_data[node_index].level + 1;
        int index = atomicAdd(&l_radix[cur_level], rot_per_level[cur_level]);
        for (int i = 0; i < rot_per_level[cur_level]; ++i) {
            g_input[index++] = make_int2(node_index, i);
        }
    }
}

__global__ void d_compute_score(
        int tree_level,
        int rot_cnt,
        int num_child,

        data_t g_data[],
        int2 g_input[],

        int *g_data_size,

        const int   g_node_offset[],
        const int   g_rot_per_level[],
        const float g_self_energy[],
        const float g_reduce_energy[],
        const float g_pm_energy[])
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int lid = threadIdx.x;

    __shared__ int   node_offset[MAX_LEVEL];
    __shared__ int   rot_per_level[MAX_LEVEL];
    extern __shared__ float self_energy[];

    if (lid < tree_level) {
        node_offset[lid] = g_node_offset[lid];
        rot_per_level[lid] = g_rot_per_level[lid];
    }
    for (int i = lid; i < rot_cnt; i += blockDim.x)
        self_energy[i] = g_self_energy[i];

    __syncthreads();

    if (id >= num_child)
        return;

    int2   input = g_input[id];
    data_t data  = g_data[input.x];
    float  score = data.g_score;

    data.level += 1;
    data.conf[data.level] = input.y;

    __compute_score(tree_level,
                    rot_cnt,
                    &data,
                    score,
                    node_offset,
                    rot_per_level,
                    self_energy,
                    g_reduce_energy,
                    g_pm_energy);

    int index = *g_data_size + id;
    g_data[index] = data;

#ifdef VERBOSE_DEBUG
    printf("[%d]Compute Score: (%d %d) to %d\n",
           id, input.x, input.y, index);
    printf("[%d]Compute Score: fscore: %.3f gscore: %.3f\n",
           id, data.f_score, data.g_score);
#endif
}

// put the generated node back to heap
__global__ void d_push_back(
        int tree_level,
        int num_child,

        heap_t g_heap[],
        data_t g_data[],
        int    g_heap_size[],
        int   *g_data_size,

        uint *g_optimal,

        int g_radix[],

        int *g_begin_index,
        int *g_begin_index2,
        int *g_output_size)
{
    int global_size = gridDim.x * blockDim.x;
    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int lid = threadIdx.x;

    int data_size  = *g_data_size;
    int data_size2 = data_size + num_child;

    int index = id - *g_begin_index;
    index = (index < 0 ? index + global_size : index);
    index += data_size;

    heap_t *heap = g_heap + d_heap_capacity*id;
    int heap_size = g_heap_size[id];

    uint optimal = (uint)-1;
    while (index < data_size2) {
        heap_t val;
        val.v = g_data[index].f_score;
        val.idx = index;

        optimal = min(optimal, flip_float(val.v));
#ifdef VERBOSE_DEBUG
        printf("[%d]: assign node (%.3f, %d) to this heap\n", id, val.v, val.idx);
#endif

        int now = ++heap_size;
        while (now > 1) {
            int next = now / 2;
            heap_t next_val = heap[next];
            if (val.v < next_val.v) {
                heap[now] = next_val;
                now = next;
            } else
                break;
        }
        heap[now] = val;

        index += global_size;
    }
    g_heap_size[id] = heap_size;

    __shared__ uint l_optimal;
    l_optimal = (uint)-1;
    __syncthreads();
    atomicMin(&l_optimal, optimal);
    __syncthreads();
    if (lid == 0)
        atomicMin(g_optimal, l_optimal);

    if (index == data_size2)
        *g_begin_index2 = id;
    if (id < tree_level)
        g_radix[id] = 0;
    if (id == 0)
        *g_output_size = 0;
}

// Shrink operation: delete node which unlikely become the answer {{{
// This can speed up calculation and do GMAC in a bounded memroy
// But it may not give the optimal answer if we apply shrink operation
// See also http://en.wikipedia.org/wiki/SMA* for detailed explaination.
__global__ void d_tagging(
        heap_t g_heap[],
        int    g_heap_size[],
        int    g_data_used[])
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;

    heap_t *heap = g_heap + d_heap_capacity*id;
    int heap_size = g_heap_size[id];

    for (int i = 1; i <= heap_size; ++i)
        g_data_used[heap[i].idx] = 1;
}

__global__ void d_scatter_data(
        int    data_size,
        int    g_data_used[],
        data_t g_old_data[],
        data_t g_new_data[],
        float  g_data_val[])
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;

    // When id == 0, we assume that it should be free.
    if (id == 0 || id >= data_size)
        return;

    if (id > 0) {
        int pos = g_data_used[id-1];
        if (g_data_used[id] != pos) {
            data_t data = g_old_data[id];
            g_new_data[pos] = data;
            g_data_val[pos] = data.f_score;
        }
    }
}

__global__ void d_reassign(
        heap_t g_heap[],
        data_t g_data[],
        int    g_heap_size[],
        int   *g_data_size)
{
    int id  = blockDim.x*blockIdx.x + threadIdx.x;
    int global_size = gridDim.x * blockDim.x;

    heap_t *heap = g_heap + d_heap_capacity*id;
    int data_size = *g_data_size;

    int cnt = 0;
    for (int i = id; i < data_size; i += global_size) {
        ++cnt;
        heap[cnt].v = g_data[i].f_score;
        heap[cnt].idx = i;
    }
    g_heap_size[id] = cnt;
}

void __print_data(int data_size)
{
    puts(">>> Print heap data <<<");

    printf("data size: %d\n", data_size);
    for (int i = 0; i < data_size; ++i) {
        data_t t = copy_from_device(d_data+i);
        printf("[%d]fscore: %.3f, level: %d\n", i, t.f_score, t.level);
    }

    puts(">>> End heap data <<<");
}

void shrink(int *__data_size, int *__num_child)
{
    int data_size = *__data_size + *__num_child;
    if (data_size + data_reserved > data_capacity || data_size > max_data) {
        if (shrink_ratio >= 0.99) {
            puts("  >>> GPU run out of memory! Running Failed");
            exit(EXIT_FAILURE);
        }
        puts("  >>> GPU run out of memory! Rescan to free memory");
        *__num_child = 0;

        if (!d_data_val)
            CHK_CUDA(hipMalloc(&d_data_val, data_capacity * sizeof(float)));
        if (!d_data_used)
            CHK_CUDA(hipMalloc(&d_data_used, data_capacity * sizeof(int)));
        if (!d_data2)
            CHK_CUDA(hipMalloc(&d_data2, data_capacity * sizeof(data_t)));

        // puts("OK0");fflush(stdout);

        CHK_CUDA(hipMemset(d_data_used, 0, data_size * sizeof(int)));
        d_tagging<<<num_block, num_local>>>(
                d_heap,
                d_heap_size,
                d_data_used);

        // puts("OK1");fflush(stdout);

        thrust::device_ptr<int> dev_used(d_data_used);
        thrust::inclusive_scan(dev_used, dev_used+data_size, dev_used);
        int new_data_size = dev_used[data_size-1];

        printf("data_size: %d\nnew_data_size: %d\n", data_size, new_data_size);
        fflush(stdout);

        d_scatter_data<<<(data_size-1) / num_local + 1, num_local>>>(
                data_size,
                d_data_used,
                d_data,
                d_data2,
                d_data_val);

        std::swap<data_t *>(d_data, d_data2);

        size_t free0;
        size_t total;
        hipMemGetInfo(&free0, &total);
        printf("free: %zuMB, total: %zuMB\n", free0/1024/1024, total/1024/1024);

        thrust::device_ptr<data_t> dev_data(d_data);
        thrust::device_ptr<float>  dev_val(d_data_val);
        thrust::sort_by_key(dev_val, dev_val+new_data_size, dev_data);
        // thrust::host_vector<data_t> h_data(dev_data, dev_data + new_data_size);
        // thrust::sort(h_data.begin(), h_data.end());
        // thrust::copy(h_data.begin(), h_data.end(), dev_data);

        new_data_size *= shrink_ratio;

        float curr_throw_min = dev_val[new_data_size];
        throw_min = min(throw_min, curr_throw_min);

        copy_to_device(d_data_size, new_data_size);
        *__data_size = new_data_size;
        printf("  >>> %d elements left\n", new_data_size);

        d_reassign<<<num_block, num_local>>>(
                d_heap,
                d_data,
                d_heap_size,
                d_data_size);
    }
}
// }}}

extern "C" void init_cuda()
{
    data_capacity = (gpu_memory - 100*1024*1024) / (sizeof(data_t)*(shrink_ratio == 1.f ? 1 : 3) + sizeof(heap_t));
    data_capacity = int(data_capacity / 1.2f);

    printf("data_capacity: %d\n", data_capacity);
    data_reserved = data_capacity / 12;
    printf("data_reserved: %d\n", data_reserved);
    num_block     = num_gpu_group;
    printf("num_block: %d\n", num_block);
    num_local     = num_gpu_item;
    printf("num_local: %d\n", num_local);
    num_local2    = num_gpu_item2;
    printf("num_local2: %d\n", num_local2);
    num_global    = num_block * num_local;
    printf("num_global: %d\n", num_global);
    heap_capacity = data_capacity / num_global;
    printf("heap_capacity: %d\n", heap_capacity);
    hipMemcpyToSymbol(HIP_SYMBOL(d_heap_capacity), &heap_capacity, sizeof(heap_capacity));

    int max_child = 0;
    func_reduce(max_child, rot_per_level, rot_per_level+tree_level, max);
    if (max_child >= 255) {
        printf("CUDA K* does not support more than 255 rotemar in single\n"
               "residue due to memory limit.\n\n"
               "You may want to change some data type char to short in\n"
               "source jni/astar-cuda.cu to avoid this limitation.\n"
               "But that would almost double the memory usage\n");
        exit(EXIT_FAILURE);
    }
    if (tree_level > MAX_LEVEL) {
        printf("CUDA K* does not support more than %d residues due to memory"
               "bound limit.\n\n"
               "You may want to change constant in jni/astar.h to avoid this"
               "limitation.  But that would almost double the memory usage\n",
               MAX_LEVEL);
        exit(EXIT_FAILURE);
    }

    /*cuda_free(d_heap);*/
    /*cuda_free(d_heap);*/
    /*cuda_free(d_data);*/
    /*cuda_free(d_heap_size);*/
    /*cuda_free(d_output);*/
    /*cuda_free(d_parent);*/
    /*cuda_free(d_node_cnt);*/
    /*cuda_free(d_radix);*/
    /*cuda_free(d_input);*/
    /*cuda_free(d_data_size);*/
    /*cuda_free(d_optimal);*/
    /*cuda_free(d_output_size);*/
    /*cuda_free(d_begin_index);*/
    /*cuda_free(d_begin_index);*/
    /*cuda_free(d_node_offset);*/
    /*cuda_free(d_rot_per_level);*/
    /*cuda_free(d_self_energy);*/
    /*cuda_free(d_reduce_energy);*/
    /*cuda_free(d_pm_energy);*/

    /*cuda_free(d_data_val);*/
    /*cuda_free(d_data_used);*/
    /*cuda_free(d_data2);*/

    if (!d_heap)
        CHK_CUDA(hipMalloc(&d_heap, heap_capacity * num_global * sizeof(heap_t)));
    if (!d_data)
        CHK_CUDA(hipMalloc(&d_data, data_capacity * sizeof(data_t)));
    if (!d_heap_size)
        CHK_CUDA(hipMalloc(&d_heap_size, num_global * sizeof(int)));
    if (!d_output)
        CHK_CUDA(hipMalloc(&d_output, num_global * sizeof(data_t)));
    if (!d_parent)
        CHK_CUDA(hipMalloc(&d_parent, num_global * sizeof(int)));
    if (!d_node_cnt)
        CHK_CUDA(hipMalloc(&d_node_cnt, num_global * sizeof(int)));
    if (!d_radix)
        CHK_CUDA(hipMalloc(&d_radix, tree_level * sizeof(int)));
    if (!d_input)
        CHK_CUDA(hipMalloc(&d_input, num_global * MAX_ROTAMER * sizeof(int2)));

    if (!d_data_size)
        CHK_CUDA(hipMalloc(&d_data_size, sizeof(int)));
    if (!d_optimal)
        CHK_CUDA(hipMalloc(&d_optimal, sizeof(int)));
    if (!d_output_size)
        CHK_CUDA(hipMalloc(&d_output_size, sizeof(int)));
    if (!d_begin_index)
        CHK_CUDA(hipMalloc(&d_begin_index, sizeof(int)));
    if (!d_begin_index2)
        CHK_CUDA(hipMalloc(&d_begin_index2, sizeof(int)));

    if (!d_node_offset)
        CHK_CUDA(hipMalloc(&d_node_offset, tree_level * sizeof(int)));
    if (!d_rot_per_level)
        CHK_CUDA(hipMalloc(&d_rot_per_level, tree_level * sizeof(int)));
    if (!d_self_energy)
        CHK_CUDA(hipMalloc(&d_self_energy, MAX_ROTAMER * sizeof(float)));
    if (!d_reduce_energy)
        CHK_CUDA(hipMalloc(&d_reduce_energy, MAX_ROTAMER * MAX_ROTAMER * sizeof(float)));
    if (!d_pm_energy)
        CHK_CUDA(hipMalloc(&d_pm_energy, MAX_ROTAMER * (tree_level+1) * sizeof(float)));

    static float h_self_energy[MAX_ROTAMER];
    static float h_reduce_energy[MAX_ROTAMER*MAX_ROTAMER];
    static float h_pm_energy[MAX_ROTAMER*(MAX_LEVEL+1)];

    for (int i = 0; i < rot_cnt; ++i)
        h_self_energy[i] = reduce_energy[rot_cnt][i] + reduce_energy[i][rot_cnt];
    float *ptr;
    ptr = h_reduce_energy;
    for (int i = 0; i < rot_cnt; ++i)
        for (int j = 0; j < rot_cnt; ++j)
            *ptr++ = reduce_energy[i][j];
    ptr = h_pm_energy;
    for (int i = 0; i < rot_cnt; ++i)
        for (int j = 0; j <= tree_level; ++j)
            *ptr++ = pm_energy[i][j];

    printf("node offset:\n");
    for (int i = 0; i < tree_level; ++i)
        printf("%d ", node_offset[i]);
    printf("\nrot_per_level:\n");
    for (int i = 0; i < tree_level; ++i)
        printf("%d ", rot_per_level[i]);
    printf("\nself_energy:\n");
    /*
       for (int i = 0; i < rot_cnt; ++i)
       printf("%.3f ", h_self_energy[i]);
       printf("\n");
     */

    CHK_CUDA(hipMemcpy(d_node_offset,
                        node_offset,
                        tree_level * sizeof(int),
                        hipMemcpyHostToDevice));
    CHK_CUDA(hipMemcpy(d_rot_per_level,
                        rot_per_level,
                        tree_level * sizeof(int),
                        hipMemcpyHostToDevice));
    CHK_CUDA(hipMemcpy(d_self_energy,
                        h_self_energy,
                        rot_cnt * sizeof(float),
                        hipMemcpyHostToDevice));
    CHK_CUDA(hipMemcpy(d_reduce_energy,
                        h_reduce_energy,
                        rot_cnt * rot_cnt * sizeof(float),
                        hipMemcpyHostToDevice));
    CHK_CUDA(hipMemcpy(d_pm_energy,
                        h_pm_energy,
                        rot_cnt * (tree_level+1) * sizeof(float),
                        hipMemcpyHostToDevice));

    rounds = 0;
    optimal = 0;
    h_num_child = 0;
    h_output_size = 0;
    h_data_size = 0;

    pq_for(node, heap) {
        /*printf("%p\n", node);*/
        /*fflush(stdout);*/
        free(*node);
    }
    pq_init(heap);

    max_data = INT32_MAX;
    char *quota_s = getenv("KSTAR_MAX_NODES");
    if (quota_s) {
        max_data = atoi(quota_s);
        printf("Environment KSTAR_MAX_NODES is setting to %d\n", max_data);
    }

    throw_min = +INFINITY;

    printf("CUDA init finish!\n");
}

extern "C" int *astar_cuda(bool first_run)
{
    puts("\n====== GPU A* start ======");
    fflush(stdout);

    hipProfilerStart();
    if (first_run) {
        CHK_CUDA(hipMemset(d_optimal, -1, sizeof(uint)));
        d_initialize<<<num_block, num_local, rot_cnt*sizeof(float)>>>(
                tree_level,
                rot_cnt,
                d_heap,
                d_data,
                d_heap_size,
                d_data_size,
                d_optimal,
                d_output_size,
                d_begin_index,
                d_begin_index2,
                d_radix,
                d_node_offset,
                d_rot_per_level,
                d_self_energy,
                d_reduce_energy,
                d_pm_energy);
        hipDeviceSynchronize();
    }


    static data_t *h_data = NULL;
    if (h_data == NULL)
        CHK_CUDA(hipHostMalloc(&h_data, num_global * sizeof(data_t)));

#define check_return() \
    if (!pq_empty(heap) && flip_float(pq_top(heap)->f_score) <= optimal) { \
        printf("GPU best result: %.9f\n", pq_top(heap)->f_score); \
        if (pq_top(heap)->f_score > throw_min) \
            printf("!!! fscore is greater than the minimal throw element.\n" \
                   "!!! GMEC is not guaranteed:(\n"); \
        for (int i = 0; i < tree_level; ++i) { \
            curr_conf[i] = pq_top(heap)->conf[i]; \
            printf("%d ", curr_conf[i]); \
        } \
        printf("\n"); \
        free(pq_pop(heap, heap_cmp)); \
        printf("GPU native A* finish in %d ms and %d rounds\n", \
               (int)wall_time_elapsed(), rounds); \
        printf("GPU memory: %d out of %d\n", h_data_size, data_capacity); \
        puts("====== GPU A* finished ======"); \
        fflush(stdout); \
        hipProfilerStop(); \
        return curr_conf; \
    } else

    wall_time_begin();
    check_return();

    for (;;) {
        rounds++;
        // printf("Current round: %d\n", rounds);

        d_delete_min<<<num_block, num_local>>>(
                tree_level,
                h_num_child,
                d_heap,
                d_data,
                d_output,
                d_heap_size,
                d_data_size,
                d_output_size,
                d_begin_index,
                d_begin_index2,
                d_optimal,
                d_parent,
                d_radix,
                d_node_cnt,
                d_rot_per_level);

        h_num_child   = copy_from_device(d_radix+tree_level-1);
        h_output_size = copy_from_device(d_output_size);
        // printf("num_child: %d\n", h_num_child);

        // No answer
        if (!first_run && h_num_child == 0) {
            memset(curr_conf, -1, sizeof(int)*(uint)tree_level);
            printf("GPU native A* finish in %d ms and %d rounds\n",
                   (int)wall_time_elapsed(), rounds);
            printf("GPU memory: %d out of %d\n", h_data_size, data_capacity);
            puts("====== GPU A* finished ======");
            return curr_conf;
        }
        if (h_output_size > 0) {
            CHK_CUDA(hipMemcpy(h_data,
                                d_output,
                                h_output_size * sizeof(data_t),
                                hipMemcpyDeviceToHost));
            for (int i = 0; i < h_output_size; ++i) {
                data_t *node = (data_t *)malloc(sizeof(data_t));
                *node = h_data[i];
#ifdef VERBOSE_DEBUG
                printf("I reveice %.8f\n", node->f_score);
#endif
                pq_push(heap, node, heap_cmp);
            }
        }

        d_radix_sort<<<num_block, num_local>>>(
                tree_level,
                d_data,
                d_data_size,
                d_radix,
                d_parent,
                d_node_cnt,
                d_input,
                d_rot_per_level);
        int num_block2 = (h_num_child-1) / num_local2 + 1;
        d_compute_score<<<num_block2, num_local2, rot_cnt*sizeof(float)>>>(
                tree_level,
                rot_cnt,
                h_num_child,
                d_data,
                d_input,
                d_data_size,
                d_node_offset,
                d_rot_per_level,
                d_self_energy,
                d_reduce_energy,
                d_pm_energy);
        d_push_back<<<num_block, num_local>>>(
                tree_level,
                h_num_child,
                d_heap,
                d_data,
                d_heap_size,
                d_data_size,
                d_optimal,
                d_radix,
                d_begin_index,
                d_begin_index2,
                d_output_size);

        optimal = copy_from_device(d_optimal);
        CHK_CUDA(hipMemset(d_optimal, -1, sizeof(uint)));
        // printf("optimal: %.9f\n", reverse_flip_float(optimal));

        h_data_size = copy_from_device(d_data_size);
        shrink(&h_data_size, &h_num_child);
        // printf("h_data_size: %d\n", h_data_size);
        // printf("h_num_child: %d\n", h_num_child);

        check_return();
    }
}
